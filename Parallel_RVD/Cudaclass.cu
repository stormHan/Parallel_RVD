#include "hip/hip_runtime.h"


#include <iostream>
/*
	process the cuda error
*/
inline void checkCudaErrors(hipError_t err)
{
	if (hipSuccess != err)
	{
		fprintf(stderr, "CUDA Runtime API error : %s.\n", hipGetErrorString(err));
		return;
	}
}

extern "C" void runCuda(double* host_seeds_pointer, double* host_mesh_vertex_pointer,
	int* host_facet_index, int points_nb, int mesh_vertex_nb, int mesh_facet_number)
{
	double* dev_seeds_pointer;
	double* dev_mesh_vertex_pointer;
	int* dev_mesh_facet_index;

	//allocate the memory
	checkCudaErrors(hipMalloc((void**)&dev_seeds_pointer, sizeof(double) * points_nb * 3));
	checkCudaErrors(hipMalloc((void**)&dev_mesh_vertex_pointer, sizeof(double) * mesh_vertex_nb * 3));
	checkCudaErrors(hipMalloc((void**)&dev_mesh_facet_index, sizeof(int) * mesh_facet_number * 3));

	//pass the data from host to device
	checkCudaErrors(hipMemcpy(dev_seeds_pointer, host_seeds_pointer, sizeof(double) * points_nb * 3, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(dev_mesh_vertex_pointer, host_mesh_vertex_pointer, sizeof(double) * points_nb * 3, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(dev_mesh_facet_index, host_facet_index, sizeof(int) * points_nb * 3, hipMemcpyHostToDevice));

}